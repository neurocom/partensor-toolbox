#include "hip/hip_runtime.h"
// #include "hip/hip_runtime.h"
// #include "hip/hip_runtime.h"
// #include ""
#include "add_vectors.hpp"

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

extern "C" void cuda_vecAdd(double *A, double *B, double *C, int length)
{
    int blockSize, gridSize;
    blockSize = CUDA_VEC_BLOCK_SIZE;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)length / blockSize);

    vecAdd<<<gridSize, blockSize>>>(A, B, C, length);
}   